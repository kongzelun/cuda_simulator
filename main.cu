#include "hip/hip_runtime.h"
//#include "simulator.h"
#include "system.h"
#include "resources.h"
#include "scheduler.h"
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include "list.cu"
#include <thrust/device_vector.h>
#include <thrust/copy.h>

using namespace std;

#define LIGHT ((((float)rand())/(float) RAND_MAX) * (0.2-0.05) + 0.05)
#define MEDIUM ((((float)rand())/(float) RAND_MAX) * (0.5-0.2) + 0.2)
#define HEAVY ((((float)rand())/(float) RAND_MAX) * (0.8-0.5) + 0.5)
#define MIXED ((((float)rand())/(float) RAND_MAX) * (0.8-0.05) + 0.05)

#define TASK_MODE LIGHT

enum PER_TASK_UTILIZATION { light , medium , heavy, mixed};

#define NUMBER  100
#define PROCESSOR_NUMBER  4
#define TOTAL_UTILIZATION  0.1
#define MAX_PERIOD 200
#define MIN_PERIOD 50
#define DURATION 1000
#define OVERHEAD 0

enum SCHEDULER {FIFO , EDF, ROLE_BASED};

SCHEDULER _SCHEDULER = FIFO;

#define THREADS_PER_BLOCK 48
#define BLOCKS_PER_GRID ((NUMBER+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK)

std::list<task *> _tasks;
std::list<Processor *> _processors;

__device__ Processor** gd_processors;
__device__ task** gd_tasks;

__device__ int number_of_tasks = 0;

__device__ int _duration;
__device__ bool _preemptive;
__device__ int _overhead;
__device__ bool _sporadic;
__device__ bool _schedulibility;
__device__ int _cycle;
__device__ sched::Scheduler *_scheduler;

__device__ double total_utilization()
{
    double sum=0;
    for(int i=0; i< number_of_tasks; i++)
    {
        //task tsk = 
        sum+= gd_tasks[i]->_execution_time / gd_tasks[i]->_period;
    }
    return sum;
}
__device__ void tick()
{
    _cycle++;
}
__device__ void deadline_missed_handler()
{
    _schedulibility = false;
}
__device__ List<job *> _release()
{
   List<job *> jobs;
    for (int i=0; i< number_of_tasks; i++)
    {
        if(((_cycle - gd_tasks[i]->_phase)%gd_tasks[i]->_period ==0) || (_cycle == gd_tasks[i]->_phase))
        {
            job *tmp_job = new job(gd_tasks[i]);
            jobs.push(tmp_job);
            tmp_job->job_release(_cycle);
        }
    }
    return jobs;
}



__device__ int gcd(int a, int b) 
{ 
    if (b == 0) 
        return a; 
    return gcd(b, a % b); 
} 
  
// Returns LCM of array elements 
__device__ int findlcm(task** arr, int n) 
{ 
    // Initialize result 
    int ans = arr[0]->_period; 
  
    // ans contains LCM of arr[0], ..arr[i] 
    // after i'th iteration, 
    for (int i = 1; i < n; i++) 
        ans = (((arr[i]->_period * ans)) / 
                (gcd(arr[i]->_period, ans))); 
  
    return ans; 
} 




__device__ int hyperperiod()
{
    return findlcm(gd_tasks, number_of_tasks);
}





__global__ void kernel_run( Processor** d_processors, task** d_tasks, int no_tasks)
{
    number_of_tasks = no_tasks;
    _duration = hyperperiod();
    _cycle = 0;
    _overhead = OVERHEAD;
   while(_cycle < _duration)
    {
        List<job *> released_jobs = _release();
        _scheduler->schedule(_cycle, gd_processors, released_jobs, PROCESSOR_NUMBER);

        for (int i=0; i<PROCESSOR_NUMBER; i++)
        {
            gd_processors[i]->run(_cycle);
        }

        tick();

    }

    for (int i=0; i<PROCESSOR_NUMBER; i++)
    {
        gd_processors[i]->stop();
    }

}


int main() { 

    std::list<task *> _tasks;
    srand(0);
    double total = 0.0;

    int i=0;

    double total_utilization = TOTAL_UTILIZATION * PROCESSOR_NUMBER;

    while ((total_utilization - total) > (1/MAX_PERIOD))
    {
        int period = rand()%(200-50+1) + 50;
        int util = TASK_MODE;
        
        if(total+util > total_utilization)
        {
            util = total_utilization - total;
        }

        int execution_time = floor(util * period);

        if(execution_time < 1)
        {
            execution_time = 1;
            period = ceil(execution_time/util);
        }

        util = execution_time/period;

        total += util;
        task *tsk = new task("task"+to_string(i++), 0 ,period, period, execution_time);

        _tasks.push_back(tsk);

        //total_utilization = total;
    }
    
    for (int i=0; i< PROCESSOR_NUMBER; i++)
    {
        Processor *p = new Processor("cpu"+to_string(i), 1, 0);
        _processors.push_back(p);
    }

    switch (_SCHEDULER)
    {
    case FIFO:
        hipMalloc((void**)&_scheduler, sizeof(sched::FIFO));
        break;
    
    default:
        break;
    }

    thrust::device_vector<Processor *> d_processors(_processors.begin(), _processors.end());
    thrust::device_vector<task *> d_tasks(_tasks.begin(), _tasks.end());
    gd_processors = thrust::raw_pointer_cast( &d_processors[0] );
    gd_tasks = thrust::raw_pointer_cast( &d_tasks[0] );


    kernel_run<<<THREADS_PER_BLOCK, BLOCKS_PER_GRID>>>(gd_processors,gd_tasks, _tasks.size());


    cout << "Hello" << endl;
    return 0; }
