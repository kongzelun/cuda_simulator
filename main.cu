#include "hip/hip_runtime.h"
//#include "simulator.h"
#include "system.h"
#include "resources.h"
#include <iostream>
#include <stdlib>
#include <math.h>
#include <list>

using namespace std;

#define LIGHT rand()%(0.2-0.05+1) + 0.05;
#define MEDIUM rand()%(0.5-0.2+1) + 0.2;
#define HEAVY rand()%(0.8-0.5+1) + 0.5;
#define MIXED rand()%(0.8-0.05+1) + 0.05;

#define TASK_MODE LIGHT

enum PER_TASK_UTILIZATION { light , medium , heavy, mixed};

#define NUMBER  100
#define PROCESSOR_NUMBER  4
#define TOTAL_UTILIZATION  0.1
#define MAX_PERIOD 200
#define MIN_PERIOD 50



#define THREADS_PER_BLOCK 48
#define BLOCKS_PER_GRID ((NUMBER+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK)

__global__ void kernel_run()
{
   while(_cycle < _duration)
    {
        std::list<job *> released_jobs = _release();
        _scheduler->run(_cycle, _processors, released_jobs);

        for (Processor *p: _processors)
        {
            p->run(_cycle);
        }

        tick();

    }

    for (Processor *p: _processors)
    {
        p->stop();
    }

}


int main() { 

    std::list<task *> _tasks;
    srand(0);
    double total = 0.0;

    int i=0;

    double total_utilization = TOTAL_UTILIZATION * PROCESSOR_NUMBER;

    while ((total_utilization - total) > (1/MAX_PERIOD))
    {
        int period = rand()%(200-50+1) + 50;
        int util = TASK_MODE;
        
        if(total+util > total_utilization)
        {
            util = total_utilization - total;
        }

        int execution_time = floor(util * period);

        if(execution_time < 1)
        {
            execution_time = 1;
            period = ceil(execution_time/util);
        }

        util = execution_time/period;

        total += util;
        task *tsk = new task("task"+to_string(i++),period, period, execution_time);

        _tasks.push_back(tsk);

        //total_utilization = total;
    }
    
    kernel_run<<<THREADS_PER_BLOCK, BLOCKS_PER_GRID>>>();


    cout << "Hello" << endl;
    return 0; }