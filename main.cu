#include "hip/hip_runtime.h"
//#include "simulator.h"
#include "system.cuh"
#include "resources.cuh"
#include "scheduler.cuh"
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include "list.cu"
#include "util.h"

using namespace std;

#define LIGHT(x) (((float)hiprand_uniform(x)) * (0.2-0.05) + 0.05)
#define MEDIUM(x) (((float)hiprand_uniform(x)) * (0.5-0.2) + 0.2)
#define HEAVY(x) (((float)hiprand_uniform(x)) * (0.8-0.5) + 0.5)
#define MIXED(x) (((float)hiprand_uniform(x)) * (0.8-0.05) + 0.05)

#define TASK_MODE(x) LIGHT(x)

enum PER_TASK_UTILIZATION { light, medium, heavy, mixed };

#define NUMBER  100
#define PROCESSOR_NUMBER  4
#define TOTAL_UTILIZATION  0.1
#define MAX_PERIOD 200
#define MIN_PERIOD 50
#define DURATION 1000
#define OVERHEAD 0

enum SCHEDULER { FIFO, EDF, ROLE_BASED };

SCHEDULER _SCHEDULER = FIFO;

#define THREADS_PER_BLOCK 48
#define BLOCKS_PER_GRID ((NUMBER+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK)

//std::list<task *> _tasks;
//std::list<Processor *> _processors;

__device__ List<Processor* > *_processors;
__device__ List<task* > *_tasks;



__device__ int number_of_tasks = 0;

__device__ int _duration;
__device__ bool _preemptive;
__device__ int _overhead;
__device__ bool _sporadic;
__device__ bool _schedulibility;
__device__ int _cycle;
__device__ sched::Scheduler *_scheduler;

__device__ double total_utilization()
{
	double sum = 0;
	for (int i = 0; i< number_of_tasks; i++)
	{
		//task tsk = 
		sum += _tasks->get_index(i)->_execution_time / _tasks->get_index(i)->_period;
	}
	return sum;
}
__device__ void tick()
{
	_cycle++;
}
__device__ void deadline_missed_handler()
{
	_schedulibility = false;
}
__device__ List<job *> _release()
{
	List<job *> jobs;
	for (int i = 0; i< number_of_tasks; i++)
	{
		if (((_cycle - _tasks->get_index(i)->_phase) % _tasks->get_index(i)->_period == 0) || (_cycle == _tasks->get_index(i)->_phase))
		{
			job *tmp_job = new job(_tasks->get_index(i));
			jobs.push(tmp_job);
			tmp_job->job_release(_cycle);
		}
	}
	return jobs;
}



__device__ int gcd(int a, int b)
{
	if (b == 0)
		return a;
	return gcd(b, a % b);
}

// Returns LCM of array elements 
__device__ int findlcm(List<task *> *arr, int n)
{
	// Initialize result 
	int ans = arr->get_index(0)->_period;

	// ans contains LCM of arr[0], ..arr[i] 
	// after i'th iteration, 
	for (int i = 1; i < n; i++)
	{
		//printf("%d, ", arr->get_index(i)->_period);
		ans = (((arr->get_index(i)->_period * ans)) / (gcd(arr->get_index(i)->_period, ans)));
	}

	return ans;
}




__device__ int hyperperiod()
{
	return findlcm(_tasks, number_of_tasks);
}





__global__ void kernel_run()
{
	number_of_tasks = _tasks->size();
	_duration = hyperperiod();
	_cycle = 0;
	_overhead = OVERHEAD;
	while (_cycle < _duration)
	{
		List<job *> released_jobs = _release();
		_scheduler->schedule(_cycle, *_processors, released_jobs, PROCESSOR_NUMBER);

		for (int i = 0; i<PROCESSOR_NUMBER; i++)
		{
			_processors->get_index(i)->run(_cycle);
		}

		tick();

	}

	for (int i = 0; i<PROCESSOR_NUMBER; i++)
	{
		_processors->get_index(i)->stop();
	}

}

__global__ void build_taskset(hiprandState *my_curandstate)
{

	_processors = new List<Processor* >();
	_tasks = new List<task* >();

	//List<task *> _tasks;
	hiprand_init(1234, 0, 0, &my_curandstate[0]);
	double total = 0.0;

	int i = 0;

	int counter = 0;
	double total_utilization = TOTAL_UTILIZATION * PROCESSOR_NUMBER;


	while ((total_utilization - total) > (1 / MAX_PERIOD))
	{
		float tmp_period = hiprand_uniform(my_curandstate);
		tmp_period *= (MAX_PERIOD - MIN_PERIOD + 1);
		tmp_period += MIN_PERIOD;

		int period = (int)truncf(tmp_period);

		//printf("%d\n", period);
		double util = TASK_MODE(my_curandstate);

		//printf("%f\n", util);

		if (total + util > total_utilization)
		{
			util = total_utilization - total;
		}

		int execution_time = floor(util * period);

		//std::cout << util << std::endl;
		if (execution_time < 1)
		{
			execution_time = 1;
			period = ceil(execution_time / util);


		}

		util = (double)execution_time / period;

		total += util;
		task *tsk = new task(0, period, period, execution_time);

		

		_tasks->push(tsk);


		//total_utilization = total;

		counter++;
		/*if ((counter % 10000) == 0)
		{
			std::cout << counter << std::endl;
		}*/
	}
	//printf("here\n");

	//std::cout << "here" << std::endl;
	for (int i = 0; i< PROCESSOR_NUMBER; i++)
	{
		Processor *p = new Processor(1, 0);
		_processors->push(p);
	}

	//printf("here\n");
	// std::cout << "here" << std::endl;

}


int main() {

	hiprandState *d_state;
	hipMalloc(&d_state, sizeof(hiprandState));

	build_taskset << <1, 1 >> > (d_state);

	switch (_SCHEDULER)
	{
	case FIFO:
		hipMalloc((void**)&_scheduler, sizeof(sched::FIFO));
		break;

	default:
		break;
	}


	kernel_run << <THREADS_PER_BLOCK, BLOCKS_PER_GRID >> >();
	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipDeviceSynchronize());

	//cout << "Hello" << endl;
	getchar();
	getchar();
	return 0;
}
