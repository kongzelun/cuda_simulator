#include "hip/hip_runtime.h"
//#include "simulator.h"
#include "system.h"
#include "resources.h"
#include "scheduler.h"
#include <iostream>
#include <stdlib.h>
#include <math.h>
#include <list>
#include <thrust/device_vector.h>
#include <thrust/copy.h>

using namespace std;

#define LIGHT ((((float)rand())/(float) RAND_MAX) * (0.2-0.05) + 0.05)
#define MEDIUM ((((float)rand())/(float) RAND_MAX) * (0.5-0.2) + 0.2)
#define HEAVY ((((float)rand())/(float) RAND_MAX) * (0.8-0.5) + 0.5)
#define MIXED ((((float)rand())/(float) RAND_MAX) * (0.8-0.05) + 0.05)

#define TASK_MODE LIGHT

enum PER_TASK_UTILIZATION { light , medium , heavy, mixed};

#define NUMBER  100
#define PROCESSOR_NUMBER  4
#define TOTAL_UTILIZATION  0.1
#define MAX_PERIOD 200
#define MIN_PERIOD 50

#define SCHEDULER FIFO



#define THREADS_PER_BLOCK 48
#define BLOCKS_PER_GRID ((NUMBER+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK)

std::list<task *> _tasks;
std::list<Processor *> _processors;

thrust::device_vector<Processor *> gd_processors;
thrust::device_vector<task *> gd_tasks;

__device__ int _duration;
__device__ bool _preemptive;
__device__ int _overhead;
__device__ bool _sporadic;
__device__ bool _schedulibility;
__device__ int _cycle;
__device__ sched::Scheduler *_scheduler;

__device__ double total_utilization()
{
    double sum=0;
    for ( task tsk : gd_tasks)
    {
        sum+= tsk._execution_time / tsk._period;
    }
    return sum;
}
__device__ void tick()
{
    _cycle++;
}
__device__ void deadline_missed_handler()
{
    _schedulibility = false;
}
__device__ std::list<job *> _release()
{
    std::list<job *> jobs;
    for (task tsk : _tasks)
    {
        if(((_cycle - tsk._phase)%tsk._period ==0) || (_cycle == tsk._phase))
        {
            job *tmp_job = new job(tsk);
            jobs.push_back(tmp_job);
            tmp_job->release(_cycle);
        }
    }
    return jobs;
}

__global__ void kernel_run( thrust::device_vector<Processor *> d_processors, thrust::device_vector<task *> d_tasks)
{
   while(_cycle < _duration)
    {
        std::list<job *> released_jobs = _release();
        _scheduler->schedule(_cycle, _processors, released_jobs);

        for (Processor *p: _processors)
        {
            p->run(_cycle);
        }

        tick();

    }

    for (Processor *p: _processors)
    {
        p->stop();
    }

}


int main() { 

    std::list<task *> _tasks;
    srand(0);
    double total = 0.0;

    int i=0;

    double total_utilization = TOTAL_UTILIZATION * PROCESSOR_NUMBER;

    while ((total_utilization - total) > (1/MAX_PERIOD))
    {
        int period = rand()%(200-50+1) + 50;
        int util = TASK_MODE;
        
        if(total+util > total_utilization)
        {
            util = total_utilization - total;
        }

        int execution_time = floor(util * period);

        if(execution_time < 1)
        {
            execution_time = 1;
            period = ceil(execution_time/util);
        }

        util = execution_time/period;

        total += util;
        task *tsk = new task("task"+to_string(i++), 0 ,period, period, execution_time);

        _tasks.push_back(tsk);

        //total_utilization = total;
    }
    
    for (int i=0; i< PROCESSOR_NUMBER; i++)
    {
        Processor *p = new Processor("cpu"+to_string(i), 1, 0);
        _processors.push_back(p);
    }

    switch (SCHEDULER)
    {
    case FIFO:
        hipMalloc((void**)&_scheduler, sizeof(sched::FIFO));
        break;
    
    default:
        break;
    }

    thrust::device_vector<Processor *> d_processors(_processors.begin(), _processors.end());
    thrust::device_vector<task *> d_tasks(_tasks.begin(), _tasks.end());
    gd_processors = d_processors;
    gd_tasks = d_tasks;

    kernel_run<<<THREADS_PER_BLOCK, BLOCKS_PER_GRID>>>(d_processors,d_tasks);


    cout << "Hello" << endl;
    return 0; }
